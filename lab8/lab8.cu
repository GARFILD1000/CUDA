#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <stdio.h>
#include <hipblas.h>

#define CUDA_CHECK_RETURN(value) {\
	hipError_t _m_cudaStat = value;\
	if(_m_cudaStat != hipSuccess) {\
		fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
		exit(1);\
	}\
}

struct saxpy_functor 
{
	const float a;
	saxpy_functor(float _a) : a(_a) {}
	__host__ __device__ float operator()(float x, float y) 
	{
		return a * x + y;
	}
};

void saxpy(float a, thrust::device_vector<float>& x, thrust::device_vector<float>& y) 
{
	saxpy_functor func(a);
	thrust::transform(x.begin(), x.end(), y.begin(), y.begin(), func);
}

void print_array(float *data1, float *data2, int num_elem, const char *prefix) 
{
	printf("\n%s", prefix);
	for(int i = 0; i < num_elem; i++)
		printf("\n%d: 2.4%f 2.4%f ", i + 1, data1[i], data2[i]);
}


//compile it that way: 
//nvcc lab8.cu -lcublas -o lab8
int main() 
{
	hipEvent_t start, stop;
	float elapsedTime;
	long vectorSize = 1 << 24;
	float alpha = 2.0f;
	
	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));

	thrust::host_vector<float> h1(vectorSize);
	thrust::host_vector<float> h2(vectorSize);
	thrust::sequence(h1.begin(), h1.end());
	thrust::fill(h2.begin(), h2.end(), 0.4);
	
	printf("Before Thrust SAXPY\n");
	for (int i = 1; i < vectorSize; i = i << 1){
	    printf("h1[%d] = %f\n", i, h1[i]);
	    printf("h2[%d] = %f\n", i, h2[i]);
	}
	
	thrust::device_vector<float> d1 = h1;
	thrust::device_vector<float> d2 = h2;
	
	CUDA_CHECK_RETURN(hipEventRecord(start, 0));
	
	saxpy(alpha, d1, d2);
	
	CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
	
	h2 = d2;
	h1 = d1;
	
	printf("\nTHRUST Time: %f ms\n", elapsedTime);
	
	
	printf("\nAfter Thrust SAXPY\n");
	for (int i = 1; i < vectorSize; i = i << 1){
	    printf("h1[%d] = %f\n", i, h1[i]);
	    printf("h2[%d] = %f\n", i, h2[i]);
	}

	//const int num_elem = (vectorSize);
	float *A_h, *B_h, *A_dev, *B_dev;
	
	CUDA_CHECK_RETURN(hipHostMalloc((void**)&A_h, vectorSize * sizeof(float)));
	CUDA_CHECK_RETURN(hipHostMalloc((void**)&B_h, vectorSize * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&A_dev,  vectorSize * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&B_dev,  vectorSize * sizeof(float)));
	
	for(int i = 0; i < vectorSize; i++) 
	{
		A_h[i] = (float) i;
		B_h[i] = 0.4f;
	}
	
	printf("\nBefore cuBLAS SAXPY\n");
	for (int i = 1; i < vectorSize; i = i << 1){
	    printf("h1[%d] = %f\n", i, A_h[i]);
	    printf("h2[%d] = %f\n", i, B_h[i]);
	}
	
	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);
	
	const int num_rows = vectorSize;
	const int num_cols = 1;
	const size_t elem_size = sizeof(float);
	
	hipblasSetMatrix(num_rows, num_cols, elem_size, A_h, num_rows, A_dev, num_rows);
	hipblasSetMatrix(num_rows, num_cols, elem_size, B_h, num_rows, B_dev, num_rows);
	
	const int stride = 1;
	
	CUDA_CHECK_RETURN(hipEventRecord(start, 0));
	
	hipblasSaxpy(cublas_handle, vectorSize, &alpha, A_dev, stride, B_dev, stride);
	
	CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
	
	hipblasGetMatrix(num_rows, num_cols, elem_size, A_dev, num_rows, A_h, num_rows);
	hipblasGetMatrix(num_rows, num_cols, elem_size, B_dev, num_rows, B_h, num_rows);
	
	printf("\ncuBLAS Time: %f ms\n", elapsedTime);
	
	const int default_stream = 0;
	CUDA_CHECK_RETURN(hipStreamSynchronize(default_stream));
	
	printf("\nAfter cuBLAS SAXPY\n");
	for (int i = 1; i < vectorSize; i = i << 1){
	    printf("h1[%d] = %f\n", i, A_h[i]);
	    printf("h2[%d] = %f\n", i, B_h[i]);
	}
	
	hipblasDestroy(cublas_handle);
	CUDA_CHECK_RETURN(hipHostFree(A_h));  
	CUDA_CHECK_RETURN(hipHostFree(B_h)); 
	CUDA_CHECK_RETURN(hipFree(A_dev));  
	CUDA_CHECK_RETURN(hipFree(B_dev)); 
	
	CUDA_CHECK_RETURN(hipEventDestroy(start));
	CUDA_CHECK_RETURN(hipEventDestroy(stop));	
		
	return 0;
}
